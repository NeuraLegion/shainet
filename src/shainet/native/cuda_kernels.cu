#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
extern "C" {
__global__ void softmax_rows(double* out, const double* in, int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    double sum = 0.0;
    for(int j=0;j<cols;++j){
        double e = exp(row_in[j]);
        row_out[j] = e;
        sum += e;
    }
    for(int j=0;j<cols;++j){
        row_out[j] /= sum;
    }
}

__global__ void dropout(double* out, const double* in, int rows, int cols, double drop_p, unsigned long long seed) {
    int row = blockIdx.x;
    if(row >= rows) return;
    hiprandState state;
    hiprand_init(seed + row, 0, 0, &state);
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    for(int j=0;j<cols;++j){
        double r = hiprand_uniform_double(&state);
        row_out[j] = r < drop_p ? 0.0 : row_in[j];
    }
}
}
__global__ void slice_cols(double* out, const double* in, int rows, int src_cols, int start, int len){
    int row = blockIdx.x;
    int col = threadIdx.x;
    if(row >= rows || col >= len) return;
    out[row * len + col] = in[row * src_cols + start + col];
}

__global__ void set_cols(double* out, const double* in, int rows, int dst_cols, int start, int len){
    int row = blockIdx.x;
    int col = threadIdx.x;
    if(row >= rows || col >= len) return;
    out[row * dst_cols + start + col] = in[row * len + col];
}
}
