#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
extern "C" {
__global__ void softmax_rows(double* out, const double* in, int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    double sum = 0.0;
    for(int j=0;j<cols;++j){
        double e = exp(row_in[j]);
        row_out[j] = e;
        sum += e;
    }
    for(int j=0;j<cols;++j){
        row_out[j] /= sum;
    }
}

__global__ void dropout(double* out, const double* in, int rows, int cols, double drop_p, unsigned long long seed) {
    int row = blockIdx.x;
    if(row >= rows) return;
    hiprandState state;
    hiprand_init(seed + row, 0, 0, &state);
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    for(int j=0;j<cols;++j){
        double r = hiprand_uniform_double(&state);
        row_out[j] = r < drop_p ? 0.0 : row_in[j];
    }
}

__global__ void gather_rows(double* out, const double* in, const int* ids, int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    int id = ids[row];
    const double *row_in = in + id * cols;
    double *row_out = out + row * cols;
    for(int j=0;j<cols;++j){
        row_out[j] = row_in[j];
    }
}

__global__ void row_mean_var(const double* in, double* mean, double* var,
                             int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double sum = 0.0;
    double sq_sum = 0.0;
    for(int j=0;j<cols;++j){
        double v = row_in[j];
        sum += v;
        sq_sum += v*v;
    }
    double m = sum / cols;
    mean[row] = m;
    var[row] = sq_sum / cols - m*m;
}

__global__ void apply_layer_norm(double* out, const double* in,
                                 const double* mean, const double* var,
                                 int rows, int cols, double epsilon) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    double m = mean[row];
    double denom = sqrt(var[row] + epsilon);
    for(int j=0;j<cols;++j){
        row_out[j] = (row_in[j] - m) / denom;
    }
}
  
__global__ void slice_cols(double* out, const double* in, int rows, int src_cols, int start, int len){
    int row = blockIdx.x;
    int col = threadIdx.x;
    if(row >= rows || col >= len) return;
    out[row * len + col] = in[row * src_cols + start + col];
}

__global__ void set_cols(double* out, const double* in, int rows, int dst_cols, int start, int len){
    int row = blockIdx.x;
    int col = threadIdx.x;
    if(row >= rows || col >= len) return;
    out[row * dst_cols + start + col] = in[row * len + col];
}

