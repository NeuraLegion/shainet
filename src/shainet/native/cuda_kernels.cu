
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
extern "C" {
__global__ void softmax_rows(double* out, const double* in, int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    double sum = 0.0;
    for(int j=0;j<cols;++j){
        double e = exp(row_in[j]);
        row_out[j] = e;
        sum += e;
    }
    for(int j=0;j<cols;++j){
        row_out[j] /= sum;
    }
}

__global__ void dropout(double* out, const double* in, int rows, int cols, double drop_p, unsigned long long seed) {
    int row = blockIdx.x;
    if(row >= rows) return;
    hiprandState state;
    hiprand_init(seed + row, 0, 0, &state);
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    for(int j=0;j<cols;++j){
        double r = hiprand_uniform_double(&state);
        row_out[j] = r < drop_p ? 0.0 : row_in[j];
    }
}

__global__ void gather_rows(double* out, const double* in, const int* ids, int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    int id = ids[row];
    const double *row_in = in + id * cols;
    double *row_out = out + row * cols;
    for(int j=0;j<cols;++j){
        row_out[j] = row_in[j];
    }
}
}
