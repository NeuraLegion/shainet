
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
extern "C" {
__global__ void softmax_rows(double* out, const double* in, int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    double sum = 0.0;
    for(int j=0;j<cols;++j){
        double e = exp(row_in[j]);
        row_out[j] = e;
        sum += e;
    }
    for(int j=0;j<cols;++j){
        row_out[j] /= sum;
    }
}

__global__ void dropout(double* out, const double* in, int rows, int cols, double drop_p, unsigned long long seed) {
    int row = blockIdx.x;
    if(row >= rows) return;
    hiprandState state;
    hiprand_init(seed + row, 0, 0, &state);
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    for(int j=0;j<cols;++j){
        double r = hiprand_uniform_double(&state);
        row_out[j] = r < drop_p ? 0.0 : row_in[j];
    }
}

__global__ void row_mean_var(const double* in, double* mean, double* var,
                             int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double sum = 0.0;
    double sq_sum = 0.0;
    for(int j=0;j<cols;++j){
        double v = row_in[j];
        sum += v;
        sq_sum += v*v;
    }
    double m = sum / cols;
    mean[row] = m;
    var[row] = sq_sum / cols - m*m;
}

__global__ void apply_layer_norm(double* out, const double* in,
                                 const double* mean, const double* var,
                                 int rows, int cols, double epsilon) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    double m = mean[row];
    double denom = sqrt(var[row] + epsilon);
    for(int j=0;j<cols;++j){
        row_out[j] = (row_in[j] - m) / denom;
    }
}
}
