
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>

// Device kernels
__global__ void softmax_rows_kernel(double* out, const double* in, int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    double sum = 0.0;
    for(int j=0;j<cols;++j){
        double e = exp(row_in[j]);
        row_out[j] = e;
        sum += e;
    }
    for(int j=0;j<cols;++j){
        row_out[j] /= sum;
    }
}

__global__ void relu_backward_kernel(double* output, const double* input, const double* grad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
    
    output[idx] = input[idx] > 0.0 ? grad[idx] : 0.0;
}

// Host wrapper functions
extern "C" {
void softmax_rows(double* out, const double* in, int rows, int cols) {
    softmax_rows_kernel<<<rows, 1>>>(out, in, rows, cols);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in softmax_rows: %s\n", hipGetErrorString(err));
    }
}

void relu_backward(double* output, const double* input, const double* grad, int size) {
    int threads_per_block = 256;
    int blocks = (size + threads_per_block - 1) / threads_per_block;
    
    relu_backward_kernel<<<blocks, threads_per_block>>>(output, input, grad, size);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in relu_backward: %s\n", hipGetErrorString(err));
    }
}

__global__ void dropout_kernel(double* out, const double* in, int rows, int cols, double drop_p, unsigned long long seed) {
    int row = blockIdx.x;
    if(row >= rows) return;
    hiprandState state;
    hiprand_init(seed + row, 0, 0, &state);
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    for(int j=0;j<cols;++j){
        double r = hiprand_uniform_double(&state);
        row_out[j] = r < drop_p ? 0.0 : row_in[j];
    }
}

void dropout(double* out, const double* in, int rows, int cols, double drop_p, unsigned long long seed) {
    dropout_kernel<<<rows, 1>>>(out, in, rows, cols, drop_p, seed);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in dropout: %s\n", hipGetErrorString(err));
    }
}

__global__ void gather_rows_kernel(double* out, const double* in, const int* ids, int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    int id = ids[row];
    const double *row_in = in + id * cols;
    double *row_out = out + row * cols;
    for(int j=0;j<cols;++j){
        row_out[j] = row_in[j];
    }
}

void gather_rows(double* out, const double* in, const int* ids, int rows, int cols) {
    gather_rows_kernel<<<rows, 1>>>(out, in, ids, rows, cols);
    hipDeviceSynchronize();
}

__global__ void row_mean_var_kernel(const double* in, double* mean, double* var,
                                    int rows, int cols) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double sum = 0.0;
    double sq_sum = 0.0;
    for(int j=0;j<cols;++j){
        double v = row_in[j];
        sum += v;
        sq_sum += v*v;
    }
    double m = sum / cols;
    mean[row] = m;
    var[row] = sq_sum / cols - m*m;
}

void row_mean_var(const double* in, double* mean, double* var, int rows, int cols) {
    row_mean_var_kernel<<<rows, 1>>>(in, mean, var, rows, cols);
    hipDeviceSynchronize();
}

__global__ void apply_layer_norm_kernel(double* out, const double* in,
                                        const double* mean, const double* var,
                                        int rows, int cols, double epsilon) {
    int row = blockIdx.x;
    if(row >= rows) return;
    const double *row_in = in + row * cols;
    double *row_out = out + row * cols;
    double m = mean[row];
    double denom = sqrt(var[row] + epsilon);
    for(int j=0;j<cols;++j){
        row_out[j] = (row_in[j] - m) / denom;
    }
}

void apply_layer_norm(double* out, const double* in,
                      const double* mean, const double* var,
                      int rows, int cols, double epsilon) {
    apply_layer_norm_kernel<<<rows, 1>>>(out, in, mean, var, rows, cols, epsilon);
    hipDeviceSynchronize();
}

__global__ void slice_cols_kernel(double* out, const double* in, int rows, int src_cols, int start, int len){
    int row = blockIdx.x;
    int col = threadIdx.x;
    if(row >= rows || col >= len) return;
    out[row * len + col] = in[row * src_cols + start + col];
}

void slice_cols(double* out, const double* in, int rows, int src_cols, int start, int len){
    slice_cols_kernel<<<rows, len>>>(out, in, rows, src_cols, start, len);
    hipDeviceSynchronize();
}

__global__ void set_cols_kernel(double* out, const double* in, int rows, int dst_cols, int start, int len){
    int row = blockIdx.x;
    int col = threadIdx.x;
    if(row >= rows || col >= len) return;
    out[row * dst_cols + start + col] = in[row * len + col];
}

void set_cols(double* out, const double* in, int rows, int dst_cols, int start, int len){
    set_cols_kernel<<<rows, len>>>(out, in, rows, dst_cols, start, len);
    hipDeviceSynchronize();
}

__global__ void count_token_pairs_kernel(const int* a, const int* b, const int* freq,
                                         int pair_count, int vocab_size, int* counts){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= pair_count) return;
    int offset = a[idx] * vocab_size + b[idx];
    atomicAdd(&counts[offset], freq[idx]);
}

void count_token_pairs(const int* a, const int* b, const int* freq,
                       int pair_count, int vocab_size, int* counts){
    int blocks = (pair_count + 255) / 256;
    count_token_pairs_kernel<<<blocks, 256>>>(a, b, freq, pair_count, vocab_size, counts);
    hipDeviceSynchronize();
}

__global__ void layer_norm_backward_kernel(double* d_x, double* d_gamma, double* d_beta,
                                           const double* d_out, const double* x,
                                           const double* gamma, const double* mean,
                                           const double* var, const double* norm,
                                           int rows, int cols, double epsilon) {
    int row = blockIdx.x;
    if(row >= rows) return;

    const double *x_row = x + row * cols;
    const double *dout_row = d_out + row * cols;
    const double *norm_row = norm + row * cols;
    double *dx_row = d_x + row * cols;

    double m = mean[row];
    double v = var[row];
    double denom = sqrt(v + epsilon);
    double inv = 1.0 / denom;
    double col_f = (double)cols;

    // Compute sum_dout_gamma and sum_dout_gamma_norm
    double sum_dout_gamma = 0.0;
    double sum_dout_gamma_norm = 0.0;
    for(int j = 0; j < cols; ++j) {
        double doutg = dout_row[j] * gamma[j];
        sum_dout_gamma += doutg;
        sum_dout_gamma_norm += doutg * (x_row[j] - m);

        // Accumulate gradients for gamma and beta
        atomicAdd(&d_gamma[j], dout_row[j] * norm_row[j]);
        atomicAdd(&d_beta[j], dout_row[j]);
    }

    // Compute d_x
    for(int j = 0; j < cols; ++j) {
        double xm = x_row[j] - m;
        double doutg = dout_row[j] * gamma[j];
        dx_row[j] = inv * (doutg - sum_dout_gamma/col_f - xm * inv*inv / col_f * sum_dout_gamma_norm);
    }
}

void layer_norm_backward(double* d_x, double* d_gamma, double* d_beta,
                         const double* d_out, const double* x,
                         const double* gamma, const double* mean,
                         const double* var, const double* norm,
                         int rows, int cols, double epsilon) {
    layer_norm_backward_kernel<<<rows, 1>>>(d_x, d_gamma, d_beta, d_out, x,
                                            gamma, mean, var, norm,
                                            rows, cols, epsilon);
    hipDeviceSynchronize();
}

__global__ void sum_cols_kernel(double* out, const double* in, int rows, int cols) {
    int col = blockIdx.x;
    if(col >= cols) return;

    double sum = 0.0;
    for(int i = 0; i < rows; ++i) {
        sum += in[i * cols + col];
    }
    out[col] = sum;
}

void sum_cols(double* out, const double* in, int rows, int cols) {
    sum_cols_kernel<<<cols, 1>>>(out, in, rows, cols);
    hipDeviceSynchronize();
}

__global__ void mul_row_vector_kernel(double* matrix, const double* vec, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rows * cols) return;

    int col = idx % cols;
    matrix[idx] *= vec[col];
}

void mul_row_vector(double* matrix, const double* vec, int rows, int cols) {
    int threads_per_block = 256;
    int blocks = (rows * cols + threads_per_block - 1) / threads_per_block;

    mul_row_vector_kernel<<<blocks, threads_per_block>>>(matrix, vec, rows, cols);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in mul_row_vector: %s\n", hipGetErrorString(err));
    }
}

__global__ void transpose_kernel(double* out, const double* in, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rows * cols) return;

    int row = idx / cols;
    int col = idx % cols;

    // Transpose: out[col][row] = in[row][col]
    // In row-major: out[col * rows + row] = in[row * cols + col]
    out[col * rows + row] = in[row * cols + col];
}

void transpose(double* out, const double* in, int rows, int cols) {
    int threads_per_block = 256;
    int blocks = (rows * cols + threads_per_block - 1) / threads_per_block;

    transpose_kernel<<<blocks, threads_per_block>>>(out, in, rows, cols);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in transpose: %s\n", hipGetErrorString(err));
    }
}

__global__ void sigmoid_forward_kernel(double* activations, double* derivatives, const double* linear, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
    
    double val = linear[idx];
    // Sigmoid: 1 / (1 + exp(-x))
    double exp_neg_val = exp(-val);
    double sigmoid_val = 1.0 / (1.0 + exp_neg_val);
    
    activations[idx] = sigmoid_val;
    // Sigmoid derivative: σ(x) * (1 - σ(x))
    derivatives[idx] = sigmoid_val * (1.0 - sigmoid_val);
}

void sigmoid_forward(double* activations, double* derivatives, const double* linear, int size) {
    int threads_per_block = 256;
    int blocks = (size + threads_per_block - 1) / threads_per_block;
    
    sigmoid_forward_kernel<<<blocks, threads_per_block>>>(activations, derivatives, linear, size);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in sigmoid_forward: %s\n", hipGetErrorString(err));
    }
}

__global__ void apply_gradient_kernel(double* local_grad, const double* grad, const double* derivatives, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;

    local_grad[idx] = grad[idx] * derivatives[idx];
}

void apply_gradient(double* local_grad, const double* grad, const double* derivatives, int size) {
    int threads_per_block = 256;
    int blocks = (size + threads_per_block - 1) / threads_per_block;

    apply_gradient_kernel<<<blocks, threads_per_block>>>(local_grad, grad, derivatives, size);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in apply_gradient: %s\n", hipGetErrorString(err));
    }
}

__global__ void accumulate_bias_grad_kernel(double* bias_grad, const double* local_grad, int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col >= cols) return;

    double sum = 0.0;
    for (int row = 0; row < rows; row++) {
        sum += local_grad[row * cols + col];
    }
    atomicAdd(&bias_grad[col], sum);
}

void accumulate_bias_grad(double* bias_grad, const double* local_grad, int rows, int cols) {
    int threads_per_block = 256;
    int blocks = (cols + threads_per_block - 1) / threads_per_block;

    accumulate_bias_grad_kernel<<<blocks, threads_per_block>>>(bias_grad, local_grad, rows, cols);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in accumulate_bias_grad: %s\n", hipGetErrorString(err));
    }
}

__global__ void zero_matrix_kernel(double* matrix, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;

    matrix[idx] = 0.0;
}

void zero_matrix(double* matrix, int size) {
    int threads_per_block = 256;
    int blocks = (size + threads_per_block - 1) / threads_per_block;

    zero_matrix_kernel<<<blocks, threads_per_block>>>(matrix, size);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in zero_matrix: %s\n", hipGetErrorString(err));
    }
}

__global__ void element_div_kernel(double* out, const double* a, const double* b, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) return;

    double denom = b[idx];
    out[idx] = denom == 0.0 ? 0.0 : a[idx] / denom;
}

void element_div(double* out, const double* a, const double* b, int size){
    int threads_per_block = 256;
    int blocks = (size + threads_per_block - 1) / threads_per_block;

    element_div_kernel<<<blocks, threads_per_block>>>(out, a, b, size);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in element_div: %s\n", hipGetErrorString(err));
    }
}

__global__ void softmax_backward_kernel(double* output, const double* grad, const double* softmax_out, int rows, int cols) {
    int row = blockIdx.x;
    if (row >= rows) return;
    
    const double* grad_row = grad + row * cols;
    const double* softmax_row = softmax_out + row * cols;
    double* output_row = output + row * cols;
    
    // Compute sum of softmax * grad for this row
    double sum = 0.0;
    for (int j = 0; j < cols; j++) {
        sum += softmax_row[j] * grad_row[j];
    }
    
    // Compute softmax backward: softmax * (grad - sum)
    for (int j = 0; j < cols; j++) {
        output_row[j] = softmax_row[j] * (grad_row[j] - sum);
    }
}

void softmax_backward(double* output, const double* grad, const double* softmax_out, int rows, int cols) {
    softmax_backward_kernel<<<rows, 1>>>(output, grad, softmax_out, rows, cols);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error in softmax_backward: %s\n", hipGetErrorString(err));
    }
}

} // extern "C"
